
#include <hip/hip_runtime.h>
#include <iostream>

__global__
void benes(int *x,int *table, int n)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int idx = threadIdx.x;
  int a , b;
  int level = blockIdx.x+1;

  if(level==1){
    a = x[idx*2];
    b = x[idx*2+1];
  }
  else {
    a = x[table[idx*2 + (n*blockIdx.x)]];
    b = x[table[idx*2 + (n*blockIdx.x)]];
  }  
  
  x[idx*2+(level*n)] = a;
  x[idx*2+(level*n)+1] = b;


}

int main(void){
    int *x;
    int inputSize = 16;
    int routerPerCol = inputSize /2;
    int col = (log2(inputSize)*2-1);

    int *table ;
    
    hipMallocManaged(&x, inputSize*col*sizeof(int));
    hipMallocManaged(&table, inputSize*col*sizeof(int));
    for (int i = 0; i < inputSize; i++) {
        if (i==0)
            x[i] = 1;
        else
            x[i] = x[i-1]+1;
      }
      for (int i=0; i<col * routerPerCol; i++)
          table[i] = 0;
     hipDeviceSynchronize();
     benes<<< 7, 8>>>(x, table, inputSize);
    // cudaDeviceSynchronize();
    
    hipFree(x);
}
 
 