//put C:/Users/molly/Desktop/289Q/project/main.cu
//nvcc -std=c++11 main.cu

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>

// includes, project
#include <hip/hip_runtime.h>



//constucting 8x8 benes network
//four rows, 5 columns, 20 routers total

__host__
void makeLUT(int N, int* LUT){
	int M = N;
	int even = 0;
	int odd = 1;
	int LUTsize = N*(log2((double)N)*2 - 2);
	for (int i =0; i < LUTsize/2; i+=N){
		for (int j=0; j<N; j+=M){
			for (int k =0; k<M/2; k++){
				LUT[i+j+k] = even;
				even+=2;
			}
			for (int k =M/2; k<M; k++){
				LUT[i+j+k] = odd;
				odd+=2;
			}
		} even=0; odd=1; M = M/2;
	}
	for (int x=LUTsize-N, i=LUTsize/2; i<LUTsize;i+=N, x-=N){
		for(int j=0; j<N; j++){
			int newIndex = LUT[x+j-LUTsize/2];
			LUT[newIndex + i] = j;
		}
	}
	return;
}

int createMask(int n)
{
   int r = 0;
   for (int i=0; i<n; i++)
       r |= 1 << i;

   return r;
}


__global__
void benes(int N,  char* network, int* LUT){
	  int index = blockIdx.x * blockDim.x + threadIdx.x;
	  int idx = threadIdx.x;
	  int in1, in2, in1_index, in2_index;
	  int level = blockIdx.x+1;

	  if(blockIdx.x == 0){
		in1 = network[index*2];
		in2 = network[index*2+1];
	  }
	  else {
		  in1_index = LUT[idx*2 + (blockIdx.x-1)*N];
		  in2_index = LUT[idx*2 + (blockIdx.x-1)*N + 1];
		  in1 = network[in1_index];
		  in2 = network[in2_index];
	  }  

	  network[idx*2 + (blockIdx.x+1)*N] = in1;
	  network[idx*2 + (blockIdx.x+1)*N + 1] = in2;
}



int main(int argc, char *argv[]){
	
	if (argc != 3){
		printf("Usage: %s <input.txt> <size>\n", argv[0]);
		return 1;
	}
	
	std::ifstream file(argv[1], std::ios::binary);
	if (!file) {
        printf("Could not open input file\n");
        return 1;
    }

	
	int N = atoi(argv[2]);
	
	int blockSize = N/2; 
	int numBlocks = 2*log2((double)N)-1; 
	int LUTsize = N*(log2((double)N)*2 - 2);
	
	char* network;
	hipMallocManaged(&network,N*(numBlocks+1)*sizeof(char));
	memset(network,0,N*(numBlocks+1)*sizeof(char));
	file.read(network, N*sizeof(char));
	file.close();
	
	int* LUT;
	hipMallocManaged(&LUT,LUTsize*sizeof(int));
  makeLUT(N,LUT);
  
  bool *valid;
	hipMallocManaged(&valid,N*(numBlocks)*sizeof(bool));
	memset(valid,0,N*(numBlocks+1)*sizeof(bool)); 
	for(int i = 0; i < N; i++)
		valid[i] = 1;
	
	benes<<<numBlocks,blockSize>>>(N, network, LUT);
	hipDeviceSynchronize();
	
	for (int i = 0; i < LUTsize; i++){
		if (i%N == 0) printf("\n");
		printf("%d ", LUT[i]);
	}
	printf("\n");
	
	
	for (int i = 0; i < N*(numBlocks+1); i++){
		if (i%N == 0) printf("\n");
		printf("%d ", network[i]);
	}
  printf("\n");
  
  int mask = createMask(log2((double)N));
	for (int i = N*(numBlocks-1) ; i < N*(numBlocks); i++){
		if((mask & network[i]) != i % N){
			printf("ERROR in routing\n");
			return 1; 
		}
	}

	hipFree(valid);
	hipFree(LUT);
	hipFree(network);
}
 
 