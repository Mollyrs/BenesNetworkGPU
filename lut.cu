// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>



//constucting 8x8 benes network
//four rows, 5 columns, 20 routers total


__global__
void benes(int N, int rows, int columns,  int* network){

}



int main(int argc, char *argv[]){
	int N = 16; //8x8 benes network

	int LUTsize = N*(log2((double)N)*2 - 2);

	int* LUT;
	hipMallocManaged(&LUT,LUTsize*sizeof(int));
	
	int n;
	int M = N;
	int M2 = N;
	for (int i = 0; i < LUTsize/2; i+=N){
		//printf("i: %d\n",i);
		for (int j = 0; j < N; j += M2){
			M2 = M; 
			for (int k=0; k < M; k+=2){
				//printf("mem: %d\n",i+j+k);
				printf("n: %d\n",n);
				LUT[i+j+k] = n%N;
				LUT[i+j+k+1] = n%N + M/2;
				n++;
			}
			n = n*2;
			M = N/2;
		}
		
			
	}
	
	
	
	
	//benes<<<numBlocks,blockSize>>>(N, 4, 5, network);
	//cudaDeviceSynchronize();
	
	for (int i = 0; i < LUTsize/2; i++){
		if (i%N == 0) printf("\n");
		printf("%d ", LUT[i]);
	}
	printf("\n");
}