// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>



//constucting 8x8 benes network
//four rows, 5 columns, 20 routers total

//put C:/Users/molly/Desktop/289Q/project/lut.cu
//nvcc -std=c++11 lut.cu
__global__
void benes(int N, int rows, int columns,  int* network){

}



int main(int argc, char *argv[]){
	int N = 32; //8x8 benes network

	int LUTsize = N*(log2((double)N)*2 - 2);

	int* LUT;
	hipMallocManaged(&LUT,LUTsize*sizeof(int));
	
	int M = N;
	int even = 0;
	int odd = 1;
	
	for (int i =0; i < LUTsize/2; i+=N){
		for (int j=0; j<N; j+=M){
			for (int k =0; k<M/2; k++){
				LUT[i+j+k] = even;
				even+=2;
			}
			for (int k =M/2; k<M; k++){
				LUT[i+j+k] = odd;
				odd+=2;
			}
		} even=0; odd=1; M = M/2;
	}
	
	
	//benes<<<numBlocks,blockSize>>>(N, 4, 5, network);
	//cudaDeviceSynchronize();
	
	for (int i = 0; i < LUTsize/2; i++){
		if (i%N == 0) printf("\n");
		printf("%d ", LUT[i]);
	}
	printf("\n");
}