#include "hip/hip_runtime.h"
//put C:/Users/molly/Desktop/289Q/project/main.cu
//nvcc -std=c++11 main.cu

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_cooperative_groups.h>
// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace cooperative_groups;
namespace cg = cooperative_groups;

#define FILESIZE 256



__host__
void makeLUT(int N, int* LUT){
	int M = N;
	int even = 0;
	int odd = 1;
	int LUTsize = N*(log2((double)N)*2 - 2);
	for (int i =0; i < LUTsize/2; i+=N){
		for (int j=0; j<N; j+=M){
			for (int k =0; k<M/2; k++){
				LUT[i+j+k] = even;
				even+=2;
			}
			for (int k =M/2; k<M; k++){
				LUT[i+j+k] = odd;
				odd+=2;
			}
		} even=0; odd=1; M = M/2;
	}
	for (int x=LUTsize-N, i=LUTsize/2; i<LUTsize;i+=N, x-=N){
		for(int j=0; j<N; j++){
			int newIndex = LUT[x+j-LUTsize/2];
			LUT[newIndex + i] = j;
		}
	}
	return;
}

int createMask(int n)
{
   int r = 0;
   for (int i=0; i<n; i++)
       r |= 1 << i;

   return r;
}


__global__
void benes(int N, int block, char* network, int* LUT, volatile int* valid, int mask, char* data, char* output){
	int idx = threadIdx.x;
	int in1, in2, in1_index, in2_index;
	int readOffset=0;
	int fileSize = FILESIZE/2;
	int readOffsetSecondNet=fileSize;
	thread_group g = tiled_partition(this_thread_block(), 2);
		if(blockIdx.x == 0){
			while(readOffset < fileSize){
				in1 = data[idx*2 + readOffset];
				in2 = data[idx*2+1 + readOffset];
				readOffset+=N;
				//printf("Block %d produced %d %d\n", blockIdx.x, in1, in2);
				//printf("waiting for next block %d to consume\n", blockIdx.x + 1);
				while((valid[idx + (blockIdx.x+1)*(N/2)])==1);
				// if ((in1 & mask) < (in2 & mask)){
				network[idx*2 + (blockIdx.x+1)*N] = in1;  
				network[idx*2 + (blockIdx.x+1)*N + 1] = in2;
				
				// }
				// else{
				// 	network[idx*2 + (blockIdx.x+1)*N] = in2;  
				// 	network[idx*2 + (blockIdx.x+1)*N + 1] = in1;
				// }
				g.sync();
				valid[idx + (blockIdx.x+1)*(N/2)]=1;// valid[idx*2 + 1 + (blockIdx.x+1)*N]=1;
				
			}
		}
		
		else if ( blockIdx.x < block) {
			while(readOffset < fileSize){
				//printf("waiting for previous block %d to produce\n", blockIdx.x - 1);
				while((valid[idx + (blockIdx.x)*(N/2)])==0);
				in1_index = LUT[idx*2 + (blockIdx.x-1)*N];
				in2_index = LUT[idx*2 + (blockIdx.x-1)*N + 1];
				in1 = network[in1_index+(blockIdx.x)*N];
				in2 = network[in2_index+(blockIdx.x)*N];
				//printf("Block %d consumed %d %d\n", blockIdx.x, in1, in2);
				valid[idx + (blockIdx.x)*(N/2)] = 0;// valid[idx*2 + 1 + (blockIdx.x)*N] = 0;
			
				//printf("waiting for next block %d to consume\n", blockIdx.x + 1);
				while((valid[idx + (blockIdx.x+1)*(N/2)])==1);
				// if ((in1 & mask) < (in2 & mask)){
				network[idx*2 + (blockIdx.x+1)*N] = in1;
				network[idx*2 + (blockIdx.x+1)*N + 1] = in2;
				g.sync();
				// }
				// else{
				// 	network[idx*2 + (blockIdx.x+1)*N] = in2;
				// 	network[idx*2 + (blockIdx.x+1)*N + 1] = in1;  
				// }
				
				if (blockIdx.x != gridDim.x - 1 && blockIdx.x != block-1){
					g.sync();
					valid[idx + (blockIdx.x+1)*(N/2)]=1;// valid[idx*2 + 1 + (blockIdx.x+1)*N]=1;
				}
				else {
					output[idx*2 + readOffset] = network[idx*2 + (blockIdx.x+1)*N];
					output[idx*2+1 + readOffset] = network[idx*2 + (blockIdx.x+1)*N + 1];
				}
				// printf("Block %d produced %d %d\n", gridDim.x,output[idx*2 + readOffset], output[idx*2+1 + readOffset]);
				readOffset += N;
			}
		} 



		else if(blockIdx.x == block){
			while(readOffsetSecondNet < FILESIZE){
				in1 = data[idx*2 + readOffsetSecondNet];
				in2 = data[idx*2+1 + readOffsetSecondNet];
				readOffsetSecondNet+=N;
				// printf("Block %d produced %d %d\n", blockIdx.x, in1, in2);
				//printf("waiting for next block %d to consume\n", blockIdx.x + 1);
				while((valid[idx + (blockIdx.x+1)*(N/2)])==1);
				// if ((in1 & mask) < (in2 & mask)){
					network[idx*2 + (blockIdx.x+1)*N] = in1;  
					network[idx*2 + (blockIdx.x+1)*N + 1] = in2;
				// }
				// else{
				// 	network[idx*2 + (blockIdx.x+1)*N] = in2;  
				// 	network[idx*2 + (blockIdx.x+1)*N + 1] = in1;
				// }
				__syncthreads();
				// printf("Block %d produced %d %d\n", blockIdx.x, network[idx*2 + (blockIdx.x+1)*N],network[idx*2 + (blockIdx.x+1)*N+1]);
				valid[idx + (blockIdx.x+1)*(N/2)]=1;// valid[idx*2 + 1 + (blockIdx.x+1)*N]=1;
				__syncthreads();
			}
		}
		
		else{
			while(readOffsetSecondNet < FILESIZE){
				// printf("waiting for previous block %d to produce\n", blockIdx.x - 1);
				while((valid[idx + (blockIdx.x)*(N/2)])==0);
				__syncthreads();
				
				// printf("waiting for previous block %d to produce\n", blockIdx.x - 1);
				in1_index = LUT[idx*2 + ((blockIdx.x%block)-1)*N];
				in2_index = LUT[idx*2 + ((blockIdx.x%block)-1)*N + 1];
				in1 = network[in1_index+(blockIdx.x)*N];
				in2 = network[in2_index+(blockIdx.x)*N];
				
				// printf("Block %d thread %d consumed %d %d\n", blockIdx.x,threadIdx.x, in1, in2);
				valid[idx + (blockIdx.x)*(N/2)] = 0; //valid[idx*2 + 1 + (blockIdx.x)*N] = 0;
			
				//printf("waiting for next block %d to consume\n", blockIdx.x + 1);
				while((valid[idx + (blockIdx.x+1)*(N/2)])==1);
				// if ((in1 & mask) < (in2 & mask)){
					network[idx*2 + (blockIdx.x+1)*N] = in1;
					network[idx*2 + (blockIdx.x+1)*N + 1] = in2;
					// printf("Block %d produced %d %d\n", blockIdx.x, in1, in2);
				// }
				// else{
				// 	network[idx*2 + (blockIdx.x+1)*N] = in2;
				// 	network[idx*2 + (blockIdx.x+1)*N + 1] = in1;  
				// }
				//printf("Block %d produced %d %d\n", blockIdx.x, in1, in2);
				if (blockIdx.x != gridDim.x - 1){
					valid[idx + (blockIdx.x+1)*(N/2)]=1; //valid[idx*2 + 1 + (blockIdx.x+1)*N]=1;
					printf("valid:%d index:%d\n",valid[idx + (blockIdx.x+1)*N],idx + (blockIdx.x+1)*N);
				}
				else {
					output[idx*2 + readOffsetSecondNet] = network[idx*2 + (blockIdx.x+1)*N];
					output[idx*2+1 + readOffsetSecondNet] = network[idx*2 + (blockIdx.x+1)*N + 1];
				}
				// printf("HEREEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEE\n");
				readOffsetSecondNet += N;
			}
		}
}



int main(int argc, char *argv[]){
	
	if (argc != 3){
		printf("Usage: %s <input.txt> <size>\n", argv[0]);
		return 1;
	}
	
	std::ifstream file(argv[1], std::ios::binary);
	if (!file) {
        printf("Could not open input file\n");
        return 1;
    }

	
	int N = atoi(argv[2]);
	if (FILESIZE<N)
		N = FILESIZE;
	int blockSize = N/2; 
	int blocks = 2*log2((double)N)-1; 
	int b = 2*log2((double)N)-1;
	int LUTsize = N*(log2((double)N)*2 - 2);
	int numBlocks;

	if (FILESIZE <= N)
		numBlocks = blocks;
	else
		numBlocks = 2*blocks;

	char* network;
	hipMallocManaged(&network,N*(numBlocks+1)*sizeof(char));
	memset(network,0,N*(numBlocks+1)*sizeof(char));
	//file.read(network, N*sizeof(char));
	//file.close();
	
	int* LUT;
	hipMallocManaged(&LUT,LUTsize*sizeof(int));
	makeLUT(N,LUT);
	int mask = createMask(log2((double)N));
  
    int *valid;
	hipMallocManaged(&valid,(N/2)*(numBlocks)*sizeof(int));
	memset(valid,0,(N/2)*(numBlocks+1)*sizeof(int)); 
	for(int i = 0; i < N/2; i++)
		valid[i] = 1;
	
	char* data;
	hipMallocManaged(&data,FILESIZE*sizeof(char));
	memset(data,0,FILESIZE*sizeof(char));
	file.read(data, FILESIZE*sizeof(char));
	file.close();
	
	char* output;
	hipMallocManaged(&output,FILESIZE*sizeof(char));
	memset(output,0,FILESIZE*sizeof(char));

	
	
	benes<<<numBlocks,blockSize>>>(N, blocks, network, LUT, valid, mask, data, output);
	hipDeviceSynchronize();
	
	
	
	
	printf("The input is:");
	for (int i = 0; i < FILESIZE; i++){
		if (i%N == 0) printf("\n");
		printf("%d ", data[i]);
	}
	printf("\n\n");

  
	printf("The output is:");
	for (int i = 0; i < FILESIZE; i++){
		if (i%N == 0) printf("\n");
		printf("%d ", output[i]);
	}
	printf("\n");
   
	hipFree(valid);
	hipFree(LUT);
	hipFree(network);
	hipFree(data);
	hipFree(output);
}
 
 
