#include "hip/hip_runtime.h"
//put C:/Users/molly/Desktop/289Q/project/main.cu
//nvcc -std=c++11 main.cu

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>
#include <hip/hip_cooperative_groups.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace cooperative_groups;
namespace cg = cooperative_groups;
//constucting 8x8 benes network
//four rows, 5 columns, 20 routers total

__host__
void makeLUT(int N, int* LUT){
	int M = N;
	int even = 0;
	int odd = 1;
	int LUTsize = N*(log2((double)N)*2 - 2);
	for (int i =0; i < LUTsize/2; i+=N){
		for (int j=0; j<N; j+=M){
			for (int k =0; k<M/2; k++){
				LUT[i+j+k] = even;
				even+=2;
			}
			for (int k =M/2; k<M; k++){
				LUT[i+j+k] = odd;
				odd+=2;
			}
		} even=0; odd=1; M = M/2;
	}
	for (int x=LUTsize-N, i=LUTsize/2; i<LUTsize;i+=N, x-=N){
		for(int j=0; j<N; j++){
			int newIndex = LUT[x+j-LUTsize/2];
			LUT[newIndex + i] = j;
		}
	}
	return;
}

int createMask(int n)
{
   int r = 0;
   for (int i=0; i<n; i++)
       r |= 1 << i;

   return r;
}


__global__
void benes(int N,  char* network, int* LUT, volatile int* valid, int mask){
	int idx = threadIdx.x;
	int in1, in2, in1_index, in2_index;
	// int level = blockIdx.x;

	// auto g = this_thread_block();
	// thread_group tile4 = tiled_partition(g, 2);
	// // if (tile4.thread_rank()==0) 
	// printf("Hello from tile4 rank %d: rank:%d\tthreadID:%d\tblockID:%d\n",tile4.thread_rank(),this_thread_block().thread_rank(),idx,level);

			
	__syncthreads();
	while((valid[idx*2 + (blockIdx.x)*N])==0 || (valid[idx*2 + (blockIdx.x)*N+1]) == 0);
		if(blockIdx.x == 0){
			in1 = network[idx*2];
			in2 = network[idx*2+1];
			if ((in1 & mask) < (in2 & mask)){
				network[idx*2 + (blockIdx.x+1)*N] = in1;  
				network[idx*2 + (blockIdx.x+1)*N + 1] = in2;
			}
			else{
				network[idx*2 + (blockIdx.x+1)*N] = in2;  
				network[idx*2 + (blockIdx.x+1)*N + 1] = in1;
			}
			valid[idx*2] = 0;  valid[idx*2 + 1] = 0;
			valid[idx*2 + (blockIdx.x+1)*N]=1; valid[idx*2 + 1 + (blockIdx.x+1)*N]=1;
			__syncthreads();
		}
		
		else {
			in1_index = LUT[idx*2 + (blockIdx.x-1)*N];
			in2_index = LUT[idx*2 + (blockIdx.x-1)*N + 1];
			in1 = network[in1_index+(blockIdx.x-1)*N];
			in2 = network[in2_index+(blockIdx.x-1)*N];
			if ((in1 & mask) < (in2 & mask)){
				network[idx*2 + (blockIdx.x)*N] = in1;
				network[idx*2 + (blockIdx.x)*N + 1] = in2;
			}
			else{
				network[idx*2 + (blockIdx.x)*N] = in2;
				network[idx*2 + (blockIdx.x)*N + 1] = in1;  
			}
			valid[idx*2 + (blockIdx.x)*N] = 0; valid[idx*2 + 1 + (blockIdx.x)*N] = 0;
			valid[idx*2 + (blockIdx.x+1)*N]=1; valid[idx*2 + 1 + (blockIdx.x+1)*N]=1;		
		} 
	

}



int main(int argc, char *argv[]){
	
	if (argc != 3){
		printf("Usage: %s <input.txt> <size>\n", argv[0]);
		return 1;
	}
	
	std::ifstream file(argv[1], std::ios::binary);
	if (!file) {
        printf("Could not open input file\n");
        return 1;
    }

	
	int N = atoi(argv[2]);
	int blockSize = N/2; 
	int numBlocks = 2*log2((double)N)-1; 
	int LUTsize = N*(log2((double)N)*2 - 2);
	
	char* network;
	hipMallocManaged(&network,N*(numBlocks+1)*sizeof(char));
	memset(network,0,N*(numBlocks+1)*sizeof(char));
	file.read(network, N*sizeof(char));
	file.close();
	
	int* LUT;
	hipMallocManaged(&LUT,LUTsize*sizeof(int));
	makeLUT(N,LUT);
	int mask = createMask(log2((double)N));
  
    int *valid;
	hipMallocManaged(&valid,N*(numBlocks)*sizeof(int));
	memset(valid,0,N*(numBlocks+1)*sizeof(int)); 
	for(int i = 0; i < N; i++)
		valid[i] = 1;
	benes<<<numBlocks,blockSize>>>(N, network, LUT, valid, mask);
	hipDeviceSynchronize();
	
	
	printf("The input is:");
	for (int i = 0; i < N; i++){
		if (i%N == 0) printf("\n");
		printf("%d ", network[i]);
	}
	printf("\n");
	printf("The intermidiate layers are:\n");
	for (int i = N; i < N*(numBlocks-1); i++){
		if (i%N == 0) printf("\n");
		printf("%d ", network[i]);
		}
	printf("\n");
  
	for (int i = N*(numBlocks-1) ; i < N*(numBlocks); i++){
		if((mask & network[i]) != i % N){
			printf("ERROR in routing\n");
			return 1; 
		}
	}
	printf("Routing was successful!\nThe output is:\n");
	for (int i = N*(numBlocks-1); i < N*(numBlocks); i++){
		printf("%d ", network[i]);
	}
	printf("\n");
   
	hipFree(valid);
	hipFree(LUT);
	hipFree(network);
}
 
 