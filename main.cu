//put C:/Users/molly/Desktop/289Q/project/main.cu
//nvcc -std=c++11 main.cu

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>

// includes, project
#include <hip/hip_runtime.h>



//constucting 8x8 benes network
//four rows, 5 columns, 20 routers total

__host__
void makeLUT(int N, int* LUT){
	int M = N;
	int even = 0;
	int odd = 1;
	int LUTsize = N*(log2((double)N)*2 - 2);
	for (int i =0; i < LUTsize/2; i+=N){
		for (int j=0; j<N; j+=M){
			for (int k =0; k<M/2; k++){
				LUT[i+j+k] = even;
				even+=2;
			}
			for (int k =M/2; k<M; k++){
				LUT[i+j+k] = odd;
				odd+=2;
			}
		} even=0; odd=1; M = M/2;
	}
	for (int x=LUTsize-N, i=LUTsize/2; i<LUTsize;i+=N, x-=N){
		for(int j=0; j<N; j++){
			int newIndex = LUT[x+j-LUTsize/2];
			LUT[newIndex + i] = j;
		}
	}
	return;
}


__global__
void benes(int N,  char* network, int* LUT){

}



int main(int argc, char *argv[]){
	
	if (argc != 3){
		printf("Usage: %s <input.txt> <size>\n", argv[0]);
		return 1;
	}
	
	std::ifstream file(argv[1], std::ios::binary);
	if (!file) {
        printf("Could not open input file\n");
        return 1;
    }

	
	int N = atoi(argv[2]);
	
	int blockSize = N/2; 
	int numBlocks = 2*log2((double)N)-1; 
	int LUTsize = N*(log2((double)N)*2 - 2);
	
	char* network;
	hipMallocManaged(&network,N*(numBlocks+1)*sizeof(char));
	memset(network,0,N*(numBlocks+1)*sizeof(char));
	file.read(network, N*sizeof(char));
	file.close();
	
	int* LUT;
	hipMallocManaged(&LUT,LUTsize*sizeof(int));
	makeLUT(N,LUT);
	
	benes<<<numBlocks,blockSize>>>(N, network, LUT);
	hipDeviceSynchronize();
	
	for (int i = 0; i < LUTsize; i++){
		if (i%N == 0) printf("\n");
		printf("%d ", LUT[i]);
	}
	printf("\n");
	
	
	for (int i = 0; i < N*(numBlocks+1); i++){
		if (i%N == 0) printf("\n");
		printf("%d ", network[i]);
	}
	printf("\n");
}